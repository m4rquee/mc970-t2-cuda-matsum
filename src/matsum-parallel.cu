
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#define T 1024
#define T_exp 10 // T = 1 << T_exp

#define CUDACHECK(cmd) { \
  hipError_t e = cmd; \
  if (e != hipSuccess) { \
    printf("Failed: Cuda error %s:%d '%s'\n", \
      __FILE__, __LINE__, hipGetErrorString(e)); \
    exit(EXIT_FAILURE); \
  } \
}

__global__ void matrix_sum(int *A, int *B, int *C, int size) {
  int i = (blockIdx.x << T_exp) + threadIdx.x;

  // Limits the domain by setting i = (i >= size ? i - 1024 : i):
  asm(
  "{"
    ".reg .pred %p;"
    "setp.ge.s32 %p, %0, %1;" // set %p with i >= size
    "@%p sub.s32 %0, %0, 1024;" // conceptually: i = (i >= size ? i - 1024 : i)
  "}"
  : "+r"(i)
  : "r"(size));

  C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
  int *A, *B, *C;
  int *d_A, *d_B, *d_C;
  int i, j;
  double t;

  // Input
  int rows, cols, size;
  FILE *input;

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return EXIT_FAILURE;
  }

  if ((input = fopen(argv[1], "r")) == NULL) {
    fprintf(stderr, "Error: could not open file\n");
    return EXIT_FAILURE;
  }

  fscanf(input, "%d", &rows);
  fscanf(input, "%d", &cols);

  size = sizeof(int) * rows * cols;
  // Allocate memory on the host
  A = (int *)malloc(size);
  B = (int *)malloc(size);
  C = (int *)malloc(size);

  // Initialize memory
  for (i = 0; i < rows; i++) {
    for (j = 0; j < cols; j++) {
      A[i * cols + j] = B[i * cols + j] = i + j;
    }
  }

  // Copy data to device
  CUDACHECK(hipMalloc(&d_A, size));
  CUDACHECK(hipMalloc(&d_B, size));
  CUDACHECK(hipMalloc(&d_C, size));
  CUDACHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

  dim3 dimGrid(ceil((float) cols * rows / T));
  dim3 dimBlock(T);

  // Compute matrix sum on device
  // Leave only the kernel and synchronize inside the timing region!
  t = omp_get_wtime();
  matrix_sum<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, cols * rows);
  CUDACHECK(hipDeviceSynchronize());
  t = omp_get_wtime() - t;

  // Copy data back to host
  CUDACHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
  CUDACHECK(hipDeviceSynchronize());

  long long int sum = 0;

  // Keep this computation on the CPU
  for (i = 0; i < rows; i++) {
    for (j = 0; j < cols; j++) {
      sum += C[i * cols + j];
    }
  }

  fprintf(stdout, "%lli\n", sum);
  fprintf(stderr, "%lf\n", t);

  free(A); free(B); free(C);
  CUDACHECK(hipFree(d_A));
  CUDACHECK(hipFree(d_B));
  CUDACHECK(hipFree(d_C));
}
